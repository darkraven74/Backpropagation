#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <cstdio>
#include <algorithm>
#include <vector>
#include <cmath>
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include "neural_network.cuh"

#define BLOCK_SIZE 32

__device__ void matrix_mul_gpu(float* a, float* b, float* c, int s1, int s2, int s3)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < s1 && j < s3)
	{
		float sum = 0;
		for (int k = 0; k < s2; k++)
		{
			sum += a[i * s2 + k] * b[k * s3 + j];
		}
		c[i * s3 + j] = sum;
	}
}

__device__ void matrix_mul_gpu(float* a, float b, float* c, int s1, int s2)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j < s2 && i < s1)
	{
		c[i * s2 + j] = a[i * s2 + j] * b;
	}
}

__device__ void matrix_mul_diagonal_gpu(float* a, float* b, float* c, int s1)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j == 0 && i < s1)
	{
		c[i] = a[i] * b[i];
	}
}

__device__ void matrix_add_gpu(float* a, float* b, float* c, int s1, int s2)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < s1 && j < s2)
	{
		c[i * s2 + j] = a[i * s2 + j] + b[i * s2 + j];
	}
}

__device__ void matrix_transpose_gpu(float* a, float* c, int s1, int s2)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < s1 && j < s2)
	{
		c[j * s1 + i] = a[i * s2 + j];
	}
}

__device__ void matrix_func_gpu(float* a, float alpha, float a_size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i == 0 && j < a_size)
	{
		a[j] = 1.0f / (1.0f + __expf(-1.0f * a[j] * alpha));
	}
}

__device__ void matrix_func_der_gpu(float* a, float* c, float alpha, float a_size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j == 0 && i < a_size)
	{
		c[i] = a[i] * alpha * (1.0f - a[i]);
	}
}

layer::layer(int size, int inputs) : size(size), inputs(inputs)
{
	float* h_weights = (float*)malloc(inputs * size * sizeof(float));
	float* h_borders = (float*)malloc(size * sizeof(float));
	hipMalloc(&weights, inputs * size * sizeof(float));
	hipMalloc(&delta_weights, inputs * size * sizeof(float));
	hipMalloc(&outputs, size * sizeof(float));
	hipMalloc(&deltas, size * sizeof(float));
	hipMalloc(&borders, size * sizeof(float));
	srand(time(NULL));
	//weights in min_w...max_w
	float min_w = -0.2f;
	float max_w = 0.2f;
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < inputs; j++) 
		{
			h_weights[j * size + i] = ((max_w - min_w) * ((float)rand() / (float)RAND_MAX) + min_w);
		}
		h_borders[i] = ((max_w - min_w) * ((float)rand() / (float)RAND_MAX) + min_w);
	}
	hipMemcpy(weights, h_weights, inputs * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(borders, h_borders, size * sizeof(float), hipMemcpyHostToDevice);
	free(h_weights);
	free(h_borders);
}

layer::layer()
{

}

neural_network::neural_network(int inputs, int depth, int hidden_layer_size, int outputs, float learning_speed,
	float momentum, float alpha)
	: inputs(inputs), depth(depth), hidden_layer_size(hidden_layer_size),
	  outputs(outputs), learning_speed(learning_speed), momentum(momentum), alpha(alpha)
{
	init();
}

neural_network::neural_network(std::string file_name)
{
	FILE* f = fopen(file_name.c_str(), "r");
	fscanf(f, "%d %d %d %d", &inputs, &outputs, &depth, &hidden_layer_size);
	fscanf(f, "%f %f %f\n", &learning_speed, &momentum, &alpha);
	thrust::host_vector<float> coeff_h(inputs);
	for (int i = 0; i < inputs; i++)
	{
		fscanf(f, "%f ", &coeff_h[i]);
	}
	coeff = coeff_h;
	init();
	for (int i = 0; i < depth; i++)
	{
		layer l = layers[i];
		int x, y;
		fscanf(f, "%d %d\n", &x, &y);
		float* weights_h = (float*)malloc(l.inputs * l.size * sizeof(float));
		float* delta_weights_h = (float*)malloc(l.inputs * l.size * sizeof(float));
		float* outputs_h = (float*)malloc(l.size * sizeof(float));
		float* borders_h = (float*)malloc(l.size * sizeof(float));
		float* deltas_h = (float*)malloc(l.size * sizeof(float));
		for (int j = 0; j < l.inputs * l.size; j++)
		{
			fscanf(f, "%f %f ", &weights_h[j], &delta_weights_h[j]);
		}
		fscanf(f, "\n");
		for (int j = 0; j < l.size; j++)
		{
			fscanf(f, "%f %f %f ", &outputs_h[j], &borders_h[j], &deltas_h[j]);
		}
		hipMemcpy(l.weights, weights_h, l.inputs * l.size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(l.delta_weights, delta_weights_h, l.inputs * l.size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(l.outputs, outputs_h, l.size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(l.borders, borders_h, l.size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(l.deltas, deltas_h, l.size * sizeof(float), hipMemcpyHostToDevice);
		free(weights_h);
		free(delta_weights_h);
		free(outputs_h);
		free(borders_h);
		free(deltas_h);
	}
	fclose(f);
}


void neural_network::teach(std::vector<std::pair <std::vector<float>, std::vector<float> > >& tests, float error,
	int max_iterations, float max_val, float min_freq)
{
	normalize(tests, max_val, min_freq);
	clock_t time = clock();
	long long count = 0;
	float curr_error = error + 1;
	while ((curr_error > error) && (count < max_iterations))
	{
		count++;
		curr_error = 0;
		random_shuffle(tests.begin(), tests.end());
		float* tests_h = (float*)malloc(tests.size() * tests[0].first.size() * sizeof(float));
		float* tests_anwsers_h = (float*)malloc(tests.size() * tests[0].second.size() * sizeof(float));
		for (int i = 0; i < tests.size(); i++)
		{
			copy(tests[i].first.begin(), tests[i].first.end(), tests_h + i * tests[0].first.size());
			copy(tests[i].second.begin(), tests[i].second.end(), tests_anwsers_h + i * tests[0].second.size());
		}
		float* tests_d;
		float* tests_anwsers_d;
		hipMalloc(&tests_d, tests.size() * tests[0].first.size() * sizeof(float));
		hipMalloc(&tests_anwsers_d, tests.size() * tests[0].second.size() * sizeof(float));
		hipMemcpy(tests_d, tests_h, tests.size() * tests[0].first.size() * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(tests_anwsers_d, tests_anwsers_h, tests.size() * tests[0].second.size() * sizeof(float), hipMemcpyHostToDevice);
		thrust::device_vector<float> errors_d(tests.size());
		for (int i = 0; i < tests.size(); i++)
		{
			forward_pass(tests_d, i, tests[0].first.size());
			backward_pass(tests_anwsers_d, i, tests[0].second.size(), thrust::raw_pointer_cast(&errors_d[0]));
		}
		hipDeviceSynchronize();
		thrust::host_vector<float> errors_h = errors_d;
		hipDeviceSynchronize();
		for (int i = 0; i < tests.size(); i++)
		{
			curr_error += errors_h[i];
		}
		free(tests_h);
		free(tests_anwsers_h);
		hipFree(tests_d);
		hipFree(tests_anwsers_d);
		curr_error /= tests.size();
		printf("ERROR: %f        count: %lld\n", curr_error, count);
	}
	printf("ERROR: %f\n", curr_error);
	printf("\ncount: %lld\n", count);
	time = clock() - time;
	printf("time: %f\n\n", (float)time / CLOCKS_PER_SEC);
}

std::vector<float> neural_network::calculate(std::vector<float> const& input)
{
	float* input_d;
	float* ans_h = (float*)malloc(outputs * sizeof(float));
	hipMalloc(&input_d, input.size() * sizeof(float));
	hipMemcpy(input_d, &input[0], input.size() * sizeof(float), hipMemcpyHostToDevice);
	forward_pass(input_d, 0, input.size());
	layer l = layers[depth - 1];
	hipMemcpy(ans_h, l.outputs, outputs * sizeof(float), hipMemcpyDeviceToHost);
	std::vector<float> ans(ans_h, ans_h + outputs);
	hipFree(input_d);
	free(ans_h);
	return ans;
}

void neural_network::save_to_file(std::string file_name)
{
	FILE* f = fopen(file_name.c_str(), "w");
	fprintf(f, "%d %d %d %d\n", inputs, outputs, depth, hidden_layer_size);
	fprintf(f, "%f %f %f\n", learning_speed, momentum, alpha);
	thrust::host_vector<float> coeff_h = coeff;
	for (int i = 0; i < coeff_h.size(); i++)
	{
		fprintf(f, "%f ", coeff_h[i]);
	}
	fprintf(f, "\n");
	for (int i = 0; i < depth; i++)
	{
		layer l = layers[i];
		fprintf(f, "%d %d\n", l.inputs, l.size);
		float* weights_h = (float*)malloc(l.inputs * l.size * sizeof(float));
		float* delta_weights_h = (float*)malloc(l.inputs * l.size * sizeof(float));
		float* outputs_h = (float*)malloc(l.size * sizeof(float));
		float* borders_h = (float*)malloc(l.size * sizeof(float));
		float* deltas_h = (float*)malloc(l.size * sizeof(float));
		hipMemcpy(weights_h, l.weights, l.inputs * l.size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(delta_weights_h, l.delta_weights, l.inputs * l.size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(outputs_h, l.outputs, l.size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(borders_h, l.borders, l.size * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(deltas_h, l.deltas, l.size * sizeof(float), hipMemcpyDeviceToHost);
		for (int j = 0; j < l.inputs * l.size; j++)
		{
			fprintf(f, "%f %f ", weights_h[j], delta_weights_h[j]);
		}
		fprintf(f, "\n");
		for (int j = 0; j < l.size; j++)
		{
			fprintf(f, "%f %f %f ", outputs_h[j], borders_h[j], deltas_h[j]);
		}
		fprintf(f, "\n");
		free(weights_h);
		free(delta_weights_h);
		free(outputs_h);
		free(borders_h);
		free(deltas_h);
	}
	fclose(f);
}

void neural_network::init()
{
	max_dim = std::max(inputs, std::max(outputs, hidden_layer_size));
	layers.push_back(layer(inputs, 0));
	layers.push_back(layer(hidden_layer_size, inputs));
	for (int i = 2; i < depth - 1; i++)
	{
		layers.push_back(layer(hidden_layer_size, hidden_layer_size));
	}
	layers.push_back(layer(outputs, hidden_layer_size));
}

__global__ void help_forward_pass_gpu(layer* layers, float* test, float* coeff, int id, int size)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	layers[0].outputs[j] = test[id * size + j] / coeff[j];
}

__global__ void forward_pass_gpu(layer* layers, int i, float alpha)
{
	matrix_mul_gpu(layers[i - 1].outputs, layers[i].weights, layers[i].outputs, 1, layers[i].inputs, layers[i].size);
	__syncthreads();
	matrix_add_gpu(layers[i].outputs, layers[i].borders, layers[i].outputs, 1, layers[i].size);
	__syncthreads();
	matrix_func_gpu(layers[i].outputs, alpha, layers[i].size);
}

void neural_network::forward_pass(float* tests, int id, int size)
{
	dim3 block(BLOCK_SIZE, 1);
	dim3 grid(1 + size / (1 + BLOCK_SIZE), 1);
	help_forward_pass_gpu<<<grid, block>>>(thrust::raw_pointer_cast(&layers[0]), tests,
		thrust::raw_pointer_cast(&coeff[0]), id, size);
	hipDeviceSynchronize();
	block.y = BLOCK_SIZE;
	grid.x = 1 + max_dim / (1 + BLOCK_SIZE);
	for (int i = 1; i < depth; i++)
	{
		forward_pass_gpu<<<grid, block>>>(thrust::raw_pointer_cast(&layers[0]), i, alpha);
		hipDeviceSynchronize();
	}
}

__global__ void help_backward_pass_gpu(layer* layers, float* tests_anwsers, int depth, float alpha, float momentum,
	 float learning_speed, float* errors, int id, int size)
{
	errors[id] = 0;
	for (int i = 0; i < size; i++)
	{
		float curr_out = layers[depth - 1].outputs[i];
		errors[id] += (tests_anwsers[id * size + i] - curr_out) * (tests_anwsers[id * size + i] - curr_out);
		layers[depth - 1].deltas[i] = (tests_anwsers[id * size + i] - curr_out) * curr_out * (1.0 - curr_out) * alpha;  
		for (int j = 0; j < layers[depth - 1].inputs; j++)
		{
			layers[depth - 1].delta_weights[j * layers[depth - 1].size + i] = momentum *
				layers[depth - 1].delta_weights[j * layers[depth - 1].size + i] + learning_speed * layers[depth - 1].deltas[i] *
				layers[depth - 2].outputs[j]; 
		}
	}
	errors[id] /= 2;
}

__global__ void backward_pass_gpu(layer* layers, float* temp, float* temp2, float* temp3, float* temp4,
	 float alpha, float momentum, float learning_speed, int i)
{
	matrix_transpose_gpu(layers[i + 1].weights, temp, layers[i + 1].inputs, layers[i + 1].size);
	__syncthreads();
	matrix_mul_gpu(layers[i + 1].deltas, temp, temp2, 1, layers[i + 1].size, layers[i + 1].inputs);
	__syncthreads();
	matrix_func_der_gpu(layers[i].outputs, temp3, alpha, layers[i].size);
	__syncthreads();
	matrix_mul_diagonal_gpu(temp3, temp2, layers[i].deltas, layers[i + 1].inputs);
	__syncthreads();
	matrix_mul_gpu(layers[i].delta_weights, momentum, layers[i].delta_weights, layers[i].inputs, layers[i].size);
	__syncthreads();
	matrix_mul_gpu(layers[i - 1].outputs, layers[i].deltas, temp4, layers[i].inputs, 1, layers[i].size);
	__syncthreads();
	matrix_mul_gpu(temp4, learning_speed, temp4, layers[i].inputs, layers[i].size);
	__syncthreads();
	matrix_add_gpu(layers[i].delta_weights, temp4, layers[i].delta_weights, layers[i].inputs, layers[i].size);
}

__global__ void help2_backward_pass_gpu(layer* layers, int i)
{
	matrix_add_gpu(layers[i].weights, layers[i].delta_weights, layers[i].weights, layers[i].inputs, layers[i].size);
}

void neural_network::backward_pass(float* tests_anwsers, int id, int size, float* errors)
{
	dim3 block(1, 1);
	dim3 grid(1, 1);
	help_backward_pass_gpu<<<grid, block>>>(thrust::raw_pointer_cast(&layers[0]), tests_anwsers, depth, alpha,
		momentum, learning_speed, errors, id, size);
	hipDeviceSynchronize(); 
	block.x = BLOCK_SIZE;
	block.y = BLOCK_SIZE;
	grid.x = 1 + max_dim / (1 + BLOCK_SIZE);
	grid.y = 1 + max_dim / (1 + BLOCK_SIZE);
	for (int i = depth - 2; i > 0; i--)
	{
		float* temp;
		float* temp2;
		float* temp3;
		float* temp4;
		hipMalloc(&temp, outputs * hidden_layer_size * sizeof(float));
		hipMalloc(&temp2, hidden_layer_size * sizeof(float));
		hipMalloc(&temp3, hidden_layer_size * sizeof(float));
		hipMalloc(&temp4, hidden_layer_size * inputs * sizeof(float));
		backward_pass_gpu<<<grid, block>>>(thrust::raw_pointer_cast(&layers[0]),
			temp, temp2, temp3, temp4, alpha, momentum, learning_speed, i);
		hipDeviceSynchronize();
		hipFree(temp);
		hipFree(temp2);
		hipFree(temp3);
		hipFree(temp4);
	}
	for (int i = 1; i < depth; i++)
	{
		help2_backward_pass_gpu<<<grid, block>>>(thrust::raw_pointer_cast(&layers[0]), i);
		hipDeviceSynchronize();
	}
}

void neural_network::normalize(std::vector<std::pair <std::vector<float>, std::vector<float> > > const& tests,
	float max_val, float min_freq)
{
	int n = tests[0].first.size();
	coeff.resize(n, 1);
	std::vector<float> sum(n);
	std::vector<int> freq(n);
	for (int i = 0; i < tests.size(); i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (abs(tests[i].first[j]) > max_val)
			{
				freq[j]++;
				sum[j] += tests[i].first[j];
			}
		}
	}
	for (int i = 0; i < n; i++)
	{
		if (freq[i] > min_freq)
		{
			coeff[i] = 1.0f * sum[i] / freq[i];
		}
	}
}