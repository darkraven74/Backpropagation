#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <iterator>
#include <vector>
#include "neural_network.cuh"

using namespace std;

int main()
{
	vector<pair<vector<float>, vector<float> > > tests;
	ifstream train_stream("train-set");
	string line;
	while (getline(train_stream, line))
	{
		vector<float> test;
		vector<float> ans;
		float value;
		istringstream iss(line);
		iss >> value;
		ans.push_back(value);
		iss >> value;
		while (iss >> value)
		{
			test.push_back(value);
		}
		tests.push_back(make_pair(test, ans));
	}

	int inputs = 141;
	int outputs = 1;
	int depth = 3;
	int hidden_layer_size = 20;
	float learning_speed = 0.1f;
	float momentum = 0.1f;
	float alpha = 1;

	float error = 0.004f;
	int max_iterations = 10;

	float max_val = 5;
	float min_freq = 1500;

	neural_network net(inputs, depth, hidden_layer_size, outputs, learning_speed, momentum, alpha);
	//neural_network net("net.txt");

	net.teach(tests, error, max_iterations, max_val, min_freq);
	//net.save_to_file("net.txt");

	freopen("results", "w", stdout);
	ifstream test_stream("test-set");
	int test_id = 1;
	int error_count = 0;
	vector<int> sum(2);
	vector<int> sum_net(2);
	vector<int> sum_net_correct(2);
	vector<float> p(2);
	vector<float> r(2);
	vector<float> f1(2);
	while (getline(test_stream, line))
	{
		vector<float> test;
		vector<float> ans;
		float value;
		istringstream iss(line);
		iss >> value;
		ans.push_back(value);
		sum[(int)value]++;	
		iss >> value;
		while (iss >> value)
		{
			test.push_back(value);
		}
		vector<float> net_ans = net.calculate(test);
		float round_ans = floor(net_ans[0] + 0.5f);
		sum_net[(int)round_ans]++;	
				
		if ((int)round_ans != (int)ans[0])
		{
			printf("ERROR! test id: %d correct: %d net_output: %f\n", test_id, (int)ans[0], net_ans[0]);
			error_count++;
		}
		else
		{
			sum_net_correct[(int)round_ans]++;
			printf("OK! test id: %d correct: %d net_output: %f\n", test_id, (int)ans[0], net_ans[0]);
		}
		test_id++;
	}
	float tests_passed = 100.0f * (test_id - error_count) / test_id;
	printf("\nerrors: %d; %.2f percent of tests passed\n\n", error_count, tests_passed);
	for (int i = 0; i < 2; i++)
	{
		p[i] = 1.0f * sum_net_correct[i] / sum_net[i];
		r[i] = 1.0f * sum_net_correct[i] / sum[i];
		f1[i] = 1.0f * ((2.0f * p[i] * r[i]) / (p[i] + r[i]));
	}
	printf("p[0]: %f r[0]: %f \n", p[0], r[0]);
	printf("p[1]: %f r[1]: %f \n", p[1], r[1]);
	printf("f1[0]: %f f1[1]: %f\n", f1[0], f1[1]);
	return 0;
}